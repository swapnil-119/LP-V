
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }

}

void addCPU(int* A, int*B, int*C, int N)
{
    for(int i=0; i<N; i++)
    {
        C[i]=A[i]+B[i];
    }
}

void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N = 10;
    int* A = new int[N];
    int* B = new int[N];
    int* C = new int[N];
    int* D = new int[N];

    size_t vectorBytes = N * sizeof(int);

    initialize(A, N);
    initialize(B, N);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;

    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    float gpu_elapsed_time;
    hipEvent_t gpu_start,gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);


    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout<<"GPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;

    cout << "Addition: ";
    print(C, N);

    float cpu_elapsed_time;
    hipEvent_t cpu_start,cpu_stop;
    hipEventCreate(&cpu_start);
    hipEventCreate(&cpu_stop);
    hipEventRecord(cpu_start);

    addCPU(A,B,D,N);

    hipEventRecord(cpu_stop);
    hipEventSynchronize(cpu_stop);
    hipEventElapsedTime(&cpu_elapsed_time, cpu_start, cpu_stop);
    hipEventDestroy(cpu_start);
    hipEventDestroy(cpu_stop);

    cout<<"CPU Elapsed time is: "<<cpu_elapsed_time<<" milliseconds"<<endl;

    cout << "Addition: ";
    print(D, N);

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] D;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}